#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <iostream>
#include "../../include/threshold/niblack.h"
#include "../../include/common/chunkedExecutor.h"

/*

    based one: https://craftofcoding.wordpress.com/2021/09/30/thresholding-algorithms-niblack-local/

*/

template <typename dtype>
__global__ void niblack_kernel_2d(dtype* image, float* output, float weight, int rows, int cols,
                                  int idz, int rows_kernel, int cols_kernel) {

  //threads
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int idy = blockIdx.y * blockDim.y + threadIdx.y;

  if (idx < rows && idy < cols) {
    //mean value
    float mean = 0;

    //standard deviation
    float standard_deviation = 0;

    //get the mean value
    get_mean_kernel_2d(image + idz * rows * cols, &mean, idx, idy, rows, cols, rows_kernel,
                       cols_kernel);

    //get the standard deviation
    get_std_kernel_2d(image + idz * rows * cols, mean, &standard_deviation, idx, idy, rows, cols,
                      rows_kernel, cols_kernel);

    //apply niblack threshold: T_{niblack} (i,j) = mean(i,j) - w * std(i,j)
    //threshold value
    float T_niblack = mean - weight * standard_deviation;

    if (image[idz * rows * cols + idx * cols + idy] > T_niblack) {
      output[idz * rows * cols + idx * cols + idy] = 255;

      return;
    }

    output[idz * rows * cols + idx * cols + idy] = 0;
  }
}

template <typename dtype>
__global__ void niblack_kernel_3d(dtype* image, float* output, float weight, int rows, int cols,
                                  int depth, int rows_kernel, int cols_kernel, int depth_kernel) {

  //threads
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int idy = blockIdx.y * blockDim.y + threadIdx.y;
  const int idz = blockIdx.z * blockDim.z + threadIdx.z;

  if (idx < rows && idy < cols && idz < depth) {

    //mean value
    float mean = 0;

    //standard deviation
    float standard_deviation = 0;

    //get the mean value
    get_mean_kernel_3d(image, &mean, idx, idy, idz, rows, cols, depth, rows_kernel, cols_kernel,
                       depth_kernel);

    get_std_kernel_3d(image, mean, &standard_deviation, idx, idy, idz, rows, cols, depth,
                      rows_kernel, cols_kernel, depth_kernel);

    //apply niblack threshold: T_{niblack} (i,j,k) = mean(i,j,k) - w * std(i,j,k)
    //threshold value
    float T_niblack = mean - weight * standard_deviation;

    if (image[idz * rows * cols + idx * cols + idy] > T_niblack) {
      output[idz * rows * cols + idx * cols + idy] = 255;

      return;
    }

    output[idz * rows * cols + idx * cols + idy] = 0;
  }
}

template __global__ void niblack_kernel_2d<int>(int* image, float* output, float weight, int rows,
                                                int cols, int idz, int rows_kernel,
                                                int cols_kernel);
template __global__ void niblack_kernel_2d<float>(float* image, float* output, float weight,
                                                  int rows, int cols, int idz, int rows_kernel,
                                                  int cols_kernel);

template __global__ void niblack_kernel_3d<int>(int* image, float* output, float weight, int rows,
                                                int cols, int depth, int rows_kernel,
                                                int cols_kernel, int depth_kernel);
template __global__ void niblack_kernel_3d<float>(float* image, float* output, float weight,
                                                  int rows, int cols, int depth, int rows_kernel,
                                                  int cols_kernel, int depth_kernel);

template <typename dtype>
void niblack_threshold(dtype* image, float* output, float weight, int rows, int cols, int depth,
                       int rows_kernel, int cols_kernel, int depth_kernel) {

  dtype* dev_image;
  float* dev_output;

  hipMalloc((void**)&dev_image, rows * cols * depth * sizeof(dtype));
  hipMalloc((void**)&dev_output, rows * cols * depth * sizeof(float));

  hipMemcpy(dev_image, image, rows * cols * depth * sizeof(dtype), hipMemcpyHostToDevice);

  if (depth_kernel == 1) {

    dim3 blockSize(32, 32);
    dim3 gridSize((rows + blockSize.y - 1) / blockSize.y, (cols + blockSize.x - 1) / blockSize.x);

    //auto start = std::chrono::high_resolution_clock::now();

    for (int idz = 0; idz < depth; ++idz) {
      niblack_kernel_2d<<<gridSize, blockSize>>>(dev_image, dev_output, weight, rows, cols, idz,
                                                 rows_kernel, cols_kernel);

      hipDeviceSynchronize();
    }

    //auto end = std::chrono::high_resolution_clock::now();
    //std::chrono::microseconds duration =
        //std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    //std::cout << "Elapsed time: " << duration.count() << " microseconds" << std::endl;

  }

  else {

    dim3 blockSize(8, 8, 8);
    dim3 gridSize((rows + blockSize.y - 1) / blockSize.y, (cols + blockSize.x - 1) / blockSize.x,
                  (depth + blockSize.z - 1) / blockSize.z);

    //auto start = std::chrono::high_resolution_clock::now();

    niblack_kernel_3d<<<gridSize, blockSize>>>(dev_image, dev_output, weight, rows, cols, depth,
                                               rows_kernel, cols_kernel, depth_kernel);

    hipDeviceSynchronize();

    //auto end = std::chrono::high_resolution_clock::now();
    //std::chrono::microseconds duration =
        //std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    //std::cout << "Elapsed time: " << duration.count() << " microseconds" << std::endl;
  }

  hipMemcpy(output, dev_output, rows * cols * depth * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(dev_image);
  hipFree(dev_output);
}

template void niblack_threshold<float>(float* image, float* output, float weight, int rows,
                                       int cols, int depth, int rows_kernel, int cols_kernel,
                                       int depth_kernel);
template void niblack_threshold<int>(int* image, float* output, float weight, int rows, int cols,
                                     int depth, int rows_kernel, int cols_kernel, int depth_kernel);
template void niblack_threshold<unsigned int>(unsigned int* image, float* output, float weight,
                                              int rows, int cols, int depth, int rows_kernel,
                                              int cols_kernel, int depth_kernel);


//chunked version
template <typename in_dtype, typename out_dtype>
void niblackThreshold3DGPU(in_dtype* hostImage, out_dtype* hostOutput, int xsize, int ysize, int zsize, int flag_verbose,
                     int nx, int ny, int nz, float weight)
{
  in_dtype* deviceImage;
  out_dtype* deviceOutput;
  unsigned int size = xsize * ysize * zsize;

  hipMalloc((void**)&deviceImage, size * sizeof(in_dtype));
  hipMalloc((void**)&deviceOutput, size * sizeof(out_dtype));

  hipMemcpy(deviceImage, hostImage, size * sizeof(in_dtype), hipMemcpyHostToDevice);

  dim3 block(8, 8, 8);

  if (zsize == 1)
  {
    block = dim3(32, 32, 1);
  }

  dim3 grid((xsize + block.x - 1) / block.x, (ysize + block.y - 1) / block.y,
            (zsize + block.z - 1) / block.z);

  if (flag_verbose==1) {
    printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
    printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);
  }
  niblack_kernel_3d<<<grid, block>>>(deviceImage, deviceOutput, weight, xsize, ysize, zsize,
                                                   nx, ny, nz);

  hipDeviceSynchronize();

  hipMemcpy(hostOutput, deviceOutput, size * sizeof(out_dtype), hipMemcpyDeviceToHost);

  hipFree(deviceImage);
  hipFree(deviceOutput);

}

// Explicit instantiation for float
template void niblackThreshold3DGPU<float, float>(float* hostImage, float* hostOutput,
                                            int xsize, int ysize, int zsize, int flag_verbose,
                                            int nx, int ny, int nz, float weight);

template void niblackThreshold3DGPU<int, float>(int* hostImage, float* hostOutput,
                                            int xsize, int ysize, int zsize, int flag_verbose,
                                            int nx, int ny, int nz, float weight);

template void niblackThreshold3DGPU<unsigned int, float>(unsigned int* hostImage, float* hostOutput,
                                            int xsize, int ysize, int zsize, int flag_verbose,
                                            int nx, int ny, int nz, float weight);


template<typename in_dtype, typename out_dtype>
void niblackThresholdChunked(in_dtype* hostImage, out_dtype* hostOutput, int xsize, int ysize, int zsize,float weight, int type3d, int flag_verbose,
                       float gpuMemory, int ngpus, int nx, int ny, int nz)
{
  if (ngpus == 0)
  {
      throw std::runtime_error("CPU implementation is not available for anisotropicDiffusion3D. "
        "Please ensure a GPU is available to execute this function.");

  }

  else if (zsize==1 || type3d == 0 || nz == 1)
  {
    //calls 2d variant
    niblack_threshold(hostImage, hostOutput,weight,xsize,ysize,zsize,nx,ny,1);
    std::cout<<"2d variant\n";

  }

  else
  {

    int ncopies = 2;
    chunkedExecutor(niblackThreshold3DGPU<in_dtype,out_dtype>, ncopies, gpuMemory, ngpus,
                    hostImage, hostOutput, xsize, ysize, zsize, flag_verbose, nx, ny, nz,weight);

  }

}

template void niblackThresholdChunked<float, float>(float* hostImage, float* hostOutput,
                                              int xsize, int ysize, int zsize, float weight, int type3d, int flag_verbose,
                                              float gpuMemory, int ngpus, int nx, int ny, int nz);

template void niblackThresholdChunked<int, float>(int* hostImage, float* hostOutput,
                                            int xsize, int ysize, int zsize, float weight,int type3d,int flag_verbose,
                                            float gpuMemory, int ngpus, int nx, int ny, int nz);

template void niblackThresholdChunked<unsigned int, float>(unsigned int* hostImage, float* hostOutput,
                                                     int xsize, int ysize, int zsize, float weight,int type3d,int flag_verbose,
                                                     float gpuMemory, int ngpus, int nx, int ny, int nz);
