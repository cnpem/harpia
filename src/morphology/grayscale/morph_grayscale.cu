#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../../../include/common/grid_block_sizes.h"
#include "../../../include/morphology/cuda_helper.h"
#include "../../../include/morphology/morph_grayscale.h"
/**
 * @brief Perform a grayscale morphological operation (erosion or dilation) on a single pixel.
 *
 * This function applies a grayscale morphological operation (either erosion or dilation) to a specific pixel
 * in a 3D image, using a given kernel that defines the neighborhood for the operation.
 *
 * @tparam dtype The data type of the image (e.g., int, unsigned int, uint16_t, etc.).
 * @param image Pointer to the input image.
 * @param output Pointer to the output image.
 * @param xsize Width of the image (number of pixels in the x-dimension).
 * @param ysize Height of the image (number of pixels in the y-dimension).
 * @param zsize Depth of the image (number of pixels in the z-dimension).
 * @param padding_bottom Padding size added to the bottom of the image.
 * @param padding_top Padding size added to the top of the image.
 * @param centerIdx x-coordinate of the center pixel where the operation is applied.
 * @param centerIdy y-coordinate of the center pixel where the operation is applied.
 * @param centerIdz z-coordinate of the center pixel where the operation is applied.
 * @param kernel Pointer to the kernel used for the morphological operation.
 * @param kernel_xsize Width of the kernel (number of elements in the x-dimension).
 * @param kernel_ysize Height of the kernel (number of elements in the y-dimension).
 * @param kernel_zsize Depth of the kernel (number of elements in the z-dimension).
 * @param operation The morphological operation to apply (EROSION or DILATION).
 *
 * @note This implementation is based on the morphological operations 
 *       described in "Digital Image Processing, 4th Edition" by R.C. Gonzalez and R.E. Woods, 
 *       particularly in Chapter 9 (Morphological Image Processing), Section 9.8, 
 *       on pages 674-679.
 * @see R.C. Gonzalez, R.E. Woods, "Digital Image Processing," 4th Edition, Pearson, 2018.
 */
template <typename dtype>
CUDA_HOSTDEV void morph_grayscale_pixel(dtype* image, dtype* output, const int xsize,
                                        const int ysize, const int zsize, const int padding_bottom,
                                        const int padding_top, int centerIdx, int centerIdy,
                                        int centerIdz, int* kernel, int kernel_xsize,
                                        int kernel_ysize, int kernel_zsize, MorphOp operation) {
  dtype* im = image;
  int* ik = kernel;

  // Initialize auxiliary value with the central pixel
  size_t centerPixelIndex = static_cast<size_t>(centerIdz) * xsize * ysize + 
                            static_cast<size_t>(centerIdy) * xsize + 
                            static_cast<size_t>(centerIdx);
  dtype aux = im[centerPixelIndex];

  size_t index;
  int imageIdx, imageIdy, imageIdz;

  int startIdx = centerIdx - kernel_xsize / 2;
  int startIdy = centerIdy - kernel_ysize / 2;
  int startIdz = centerIdz - kernel_zsize / 2;

  for (int iz = 0; iz < kernel_zsize; iz++) {
    for (int iy = 0; iy < kernel_ysize; iy++) {
      for (int ix = 0; ix < kernel_xsize; ix++) {

        imageIdx = startIdx + ix;
        imageIdy = startIdy + iy;
        imageIdz = startIdz + iz;

        // Ignore out of bounds pixels and don't care pixels
        if (imageIdx < 0 || imageIdx > xsize - 1 || imageIdy < 0 || imageIdy > ysize - 1 ||
            imageIdz < -padding_bottom || imageIdz > zsize + padding_top - 1 || ik[ix] < 0) {
          // do nothing.
        }

        else {
          index = static_cast<size_t>(imageIdz) * xsize * ysize + 
                  static_cast<size_t>(imageIdy) * xsize + 
                  static_cast<size_t>(imageIdx);
          
          if (operation == EROSION) {
            aux = (im[index] < aux) ? im[index] : aux;  // Erosion: aux is the min value
          } else {
            aux = (im[index] > aux) ? im[index] : aux;  // Dilation: aux is the max value
          }
        }
      }
    }
  }
  output[centerPixelIndex] = aux;
}
template CUDA_HOSTDEV void morph_grayscale_pixel<unsigned int>(unsigned int*, unsigned int*,
                                                               const int, const int, const int,
                                                               const int, const int, int, int, int,
                                                               int*, int, int, int,

                                                               MorphOp);
template CUDA_HOSTDEV void morph_grayscale_pixel<int>(int*, int*, const int, const int, const int,
                                                      const int, const int, int, int, int, int*,
                                                      int, int, int, MorphOp);
template CUDA_HOSTDEV void morph_grayscale_pixel<float>(float*, float*, const int, const int,
                                                        const int, const int, const int, int, int,
                                                        int, int*, int, int, int, MorphOp);
/**
 * @brief CUDA kernel to perform a grayscale morphological operation on a 3D image.
 *
 * This kernel function is executed on the GPU, applying a morphological grayscale operation (erosion or dilation)
 * to every pixel in the image. Each thread processes a single pixel by invoking `morph_grayscale_pixel`
 * for the corresponding pixel.
 *
 * @tparam dtype The data type of the image (e.g., int, unsigned int, uint16_t, etc.).
 * @param deviceImage Pointer to the input image stored in GPU memory.
 * @param deviceOutput Pointer to the output image stored in GPU memory.
 * @param xsize Width of the image (number of pixels in the x-dimension).
 * @param ysize Height of the image (number of pixels in the y-dimension).
 * @param zsize Depth of the image (number of pixels in the z-dimension).
 * @param padding_bottom Padding size added to the bottom of the image.
 * @param padding_top Padding size added to the top of the image.
 * @param kernel Pointer to the kernel used for the morphological operation.
 * @param kernel_xsize Width of the kernel (number of elements in the x-dimension).
 * @param kernel_ysize Height of the kernel (number of elements in the y-dimension).
 * @param kernel_zsize Depth of the kernel (number of elements in the z-dimension).
 * @param operation The morphological operation to apply (EROSION or DILATION).
 *
 * @note This implementation is based on the morphological operations 
 *       described in "Digital Image Processing, 4th Edition" by R.C. Gonzalez and R.E. Woods, 
 *       particularly in Chapter 9 (Morphological Image Processing), Section 9.8, 
 *       on pages 674-679.
 * @see R.C. Gonzalez, R.E. Woods, "Digital Image Processing," 4th Edition, Pearson, 2018.
 */
template <typename dtype>
__global__ void morph_grayscale_kernel(dtype* deviceImage, dtype* deviceOutput, const int xsize,
                                       const int ysize, const int zsize, const int padding_bottom,
                                       const int padding_top, int* kernel, int kernel_xsize,
                                       int kernel_ysize, int kernel_zsize, MorphOp operation) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int idy = threadIdx.y + blockIdx.y * blockDim.y;
  int idz = threadIdx.z + blockIdx.z * blockDim.z;

  if (idx < xsize && idy < ysize && idz < zsize) {
    morph_grayscale_pixel(deviceImage, deviceOutput, xsize, ysize, zsize, padding_bottom,
                          padding_top, idx, idy, idz, kernel, kernel_xsize, kernel_ysize,
                          kernel_zsize, operation);
  }
}
template __global__ void morph_grayscale_kernel<unsigned int>(unsigned int*, unsigned int*,
                                                              const int, const int, const int,
                                                              const int, const int, int*, int, int,
                                                              int, MorphOp);
template __global__ void morph_grayscale_kernel<int>(int*, int*, const int, const int, const int,
                                                     const int, const int, int*, int, int, int,
                                                     MorphOp);
template __global__ void morph_grayscale_kernel<float>(float*, float*, const int, const int,
                                                       const int, const int, const int, int*, int,
                                                       int, int, MorphOp);

template <typename dtype>
void morph_grayscale(dtype* deviceImage, dtype* deviceOutput, const int xsize, const int ysize,
                     const int zsize, const int flag_verbose, const int padding_bottom,
                     const int padding_top, int* deviceKernel, int kernel_xsize, int kernel_ysize,
                     int kernel_zsize, MorphOp operation) {
  // Set up execution configuration
  dim3 block(BLOCK_3D, BLOCK_3D, BLOCK_3D);
  if (zsize == 1)
    block = dim3(BLOCK_2D, BLOCK_2D, 1);
  dim3 grid((xsize + block.x - 1) / block.x, (ysize + block.y - 1) / block.y,
            (zsize + block.z - 1) / block.z);

  // Check grid and block dimensions from host side
  if (flag_verbose) {
    printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
    printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);
  }

  // Device erosion/dilation
  morph_grayscale_kernel<<<grid, block>>>(deviceImage, deviceOutput, xsize, ysize, zsize,
                                          padding_bottom, padding_top, deviceKernel, kernel_xsize,
                                          kernel_ysize, kernel_zsize, operation);
  hipDeviceSynchronize();  // Assures all GPU threads are finished
}
template void morph_grayscale<unsigned int>(unsigned int*, unsigned int*, const int, const int,
                                            const int, const int, const int, const int, int*, int,
                                            int, int, MorphOp);
template void morph_grayscale<int>(int*, int*, const int, const int, const int, const int,
                                   const int, const int, int*, int, int, int, MorphOp);
template void morph_grayscale<float>(float*, float*, const int, const int, const int, const int,
                                     const int, const int, int*, int, int, int, MorphOp);

template <typename dtype>
void morph_grayscale_on_device(dtype* hostImage, dtype* hostOutput, const int xsize,
                               const int ysize, const int zsize, const int flag_verbose,
                               const int padding_bottom, const int padding_top, int* kernel,
                               int kernel_xsize, int kernel_ysize, int kernel_zsize,
                               MorphOp operation) {
  // set input dimension
  size_t size = static_cast<size_t>(xsize) * ysize * zsize;
  size_t nBytes = size * sizeof(dtype);
  size_t nBytes_padding = xsize * ysize * (padding_bottom + padding_top) * sizeof(dtype);
  size_t nBytes_input = nBytes + nBytes_padding;

  // set kenrel dimension
  size_t kernel_size = kernel_xsize * kernel_ysize * kernel_zsize;
  size_t kernel_nBytes = kernel_size * sizeof(int);

  // malloc device global memory
  dtype *deviceImage, *deviceOutput, *i_deviceImage, *i_hostImage;
  int* deviceKernel;
  CHECK(hipMalloc((dtype**)&i_deviceImage, nBytes_input));
  CHECK(hipMalloc((dtype**)&deviceOutput, nBytes));
  CHECK(hipMalloc((int**)&deviceKernel, kernel_nBytes));

  // transfer data from the host to the device
  CHECK(hipMemcpy(deviceKernel, kernel, kernel_nBytes, hipMemcpyHostToDevice));

  // transfer input + padding
  i_hostImage = hostImage - padding_bottom * xsize * ysize;

  CHECK(hipMemcpy(i_deviceImage, i_hostImage, nBytes_input, hipMemcpyHostToDevice));

  deviceImage = i_deviceImage + padding_bottom * xsize * ysize;

  // device erosion/dialation

  morph_grayscale(deviceImage, deviceOutput, xsize, ysize, zsize, flag_verbose, padding_bottom,
                  padding_top, deviceKernel, kernel_xsize, kernel_ysize, kernel_zsize, operation);

  CHECK(hipMemcpy(hostOutput, deviceOutput, nBytes, hipMemcpyDeviceToHost));

  // free host memorys

  hipFree(i_deviceImage);  
  hipFree(deviceOutput);
  hipFree(deviceKernel);
}
template void morph_grayscale_on_device<unsigned int>(unsigned int*, unsigned int*, const int,
                                                      const int, const int, const int, const int,
                                                      const int, int*, int, int, int, MorphOp);
template void morph_grayscale_on_device<int>(int*, int*, const int, const int, const int, const int,
                                             const int, const int, int*, int, int, int, MorphOp);
template void morph_grayscale_on_device<float>(float*, float*, const int, const int, const int,
                                               const int, const int, const int, int*, int, int, int,
                                               MorphOp);

template <typename dtype>
void morph_grayscale_on_host(dtype* hostImage, dtype* hostOutput, const int xsize, const int ysize,
                             const int zsize, int* kernel, int kernel_xsize, int kernel_ysize,
                             int kernel_zsize, MorphOp operation) {
  for (int idz = 0; idz < zsize; idz++) {
    for (int idy = 0; idy < ysize; idy++) {
      for (int idx = 0; idx < xsize; idx++) {

        morph_grayscale_pixel(hostImage, hostOutput, xsize, ysize, zsize, 0, 0, idx, idy, idz,
                              kernel, kernel_xsize, kernel_ysize, kernel_zsize, operation);
      }
    }
  }  // Slide over image
}
template void morph_grayscale_on_host<unsigned int>(unsigned int*, unsigned int*, const int,
                                                    const int, const int, int*, int, int, int,
                                                    MorphOp);
template void morph_grayscale_on_host<int>(int*, int*, const int, const int, const int, int*, int,
                                           int, int, MorphOp);
template void morph_grayscale_on_host<float>(float*, float*, const int, const int, const int, int*,
                                             int, int, int, MorphOp);