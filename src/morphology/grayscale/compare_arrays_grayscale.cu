#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdint>  // For float, unsigned int
#include "../../../include/common/grid_block_sizes.h"
#include "../../../include/morphology/compare_arrays_grayscale.h"
#include "../../../include/morphology/cuda_helper.h"
#include "../../../include/morphology/custom_abs.h"

/**
 * @brief Kernel function to perform pixel-wise comparison to check if two grayscale arrays are equal on the 
 * GPU. 
 * 
 * This function compares two input arrays (`deviceImage1` and `deviceImage2`) element by element. 
 * If any corresponding elements are not equal, the output flag (`deviceOutput`) is set to false.The
 *  comparison is performed in parallel using CUDA threads.
 *
 * @tparam dtype The data type of the elements in the arrays.
 * @param deviceImage1 Pointer to the first input array on the device (GPU).
 * @param deviceImage2 Pointer to the second input array on the device (GPU).
 * @param deviceOutput Pointer to the output flag on the device (GPU). Should be 
 * initialized to `1` before calling this function.
 * @param size The total number of elements (pixels) in the arrays.
 */
template <typename dtype>
__global__ void compare_arrays_grayscale_kernel(dtype* deviceImage1, dtype* deviceImage2,
                                                int* deviceOutput, const size_t size,
                                                dtype tolerance) {
  size_t index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < size) {
    if (custom_abs(deviceImage1[index] - deviceImage2[index]) > tolerance) {
      atomicAnd(deviceOutput, 0);
      /**
       * @note The atomic operation is used to set `deviceOutput` to `false` in a thread-safe 
       * manner. This ensures that only one thread modifies `deviceOutput` at a time, 
       * preventing race conditions.
       */
    }
  }
}
template __global__ void compare_arrays_grayscale_kernel<int>(int*, int*, int*, const size_t, int);
template __global__ void compare_arrays_grayscale_kernel<unsigned int>(unsigned int*, unsigned int*,
                                                                       int*, const size_t,
                                                                       unsigned int);
template __global__ void compare_arrays_grayscale_kernel<float>(float*, float*, int*, const size_t,
                                                                float);

template <typename dtype>
void compare_arrays_grayscale(dtype* deviceImage1, dtype* deviceImage2, int* deviceOutput,
                              const size_t size, const int flag_verbose) {

  // Set up execution configuration
  dim3 block(BLOCK_1D);
  dim3 grid((size + block.x - 1) / block.x);

  // Tolerance for floating-point comparison
  dtype tolerance = static_cast<dtype>(1.0E-8);

  // Check grid and block dimension from host side
  if (flag_verbose) {
    printf("grid.x %d \n", grid.x);
    printf("block.x %d \n", block.x);
  }

  // Perform subtraction on the device
  compare_arrays_grayscale_kernel<<<grid, block>>>(deviceImage1, deviceImage2, deviceOutput, size,
                                                   tolerance);
  hipDeviceSynchronize();  // Ensure all GPU threads are finished
}
template void compare_arrays_grayscale<int>(int*, int*, int*, const size_t, const int);
template void compare_arrays_grayscale<unsigned int>(unsigned int*, unsigned int*, int*, 
                                                     const size_t, const int);
template void compare_arrays_grayscale<float>(float*, float*, int*, const size_t, const int);

template <typename dtype>
void compare_arrays_grayscale_on_device(dtype* hostImage1, dtype* hostImage2, int* hostOutput,
                                        const size_t size, const int flag_verbose) {

  // Set input dimension
  size_t nBytes = size * sizeof(dtype);

  // Malloc device global memory
  dtype *deviceImage1, *deviceImage2;
  int* deviceOutput;
  CHECK(hipMalloc((dtype**)&deviceImage1, nBytes));
  CHECK(hipMalloc((dtype**)&deviceImage2, nBytes));
  CHECK(hipMalloc((int**)&deviceOutput, nBytes));

  // Transfer data from the host to the device
  CHECK(hipMemcpy(deviceImage1, hostImage1, nBytes, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(deviceImage2, hostImage2, nBytes, hipMemcpyHostToDevice));

  // Perform subtraction on the device
  compare_arrays_grayscale(deviceImage1, deviceImage2, deviceOutput, size, flag_verbose);

  // Transfer data from the device to the host
  CHECK(hipMemcpy(hostOutput, deviceOutput, nBytes, hipMemcpyDeviceToHost));

  // Free device memory
  hipFree(deviceImage1);
  hipFree(deviceImage2);
  hipFree(deviceOutput);
}
template void compare_arrays_grayscale_on_device<int>(int*, int*, int*, const size_t, const int);
template void compare_arrays_grayscale_on_device<unsigned int>(unsigned int*, unsigned int*, int*,
                                                               const size_t, const int);
template void compare_arrays_grayscale_on_device<float>(float*, float*, int*, const size_t, 
                                                        const int);

template <typename dtype>
void compare_arrays_grayscale_on_host(dtype* hostImage1, dtype* hostImage2, int* hostOutput,
                                      const size_t size) {
  *hostOutput = 1;
  dtype epsilon = 1.0E-8;  // Tolerance for floating-point comparison

  for (size_t index = 0; index < size; index++) {
    if (custom_abs(hostImage1[index] - hostImage2[index]) > epsilon) {
      *hostOutput = 0;
      return;  // Exit on first mismatch
    }
  }
}
template void compare_arrays_grayscale_on_host<int>(int*, int*, int*, const size_t);
template void compare_arrays_grayscale_on_host<unsigned int>(unsigned int*, unsigned int*, int*,
                                                             const size_t);
template void compare_arrays_grayscale_on_host<float>(float*, float*, int*, const size_t);
