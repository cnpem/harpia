#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdint>  // For uint16_t, unsigned int
#include "../../../include/common/grid_block_sizes.h"
#include "../../../include/morphology/cuda_helper.h"
#include "../../../include/morphology/geodesic_morph_grayscale.h"

/**
 * @brief Perform geodesic grayscale erosion/dilation operation for one pixel.
 *
 * This function applies geodesic morphological operations (erosion or dilation) 
 * to a single pixel in a 3D image. The function determines the new pixel value based on the 
 * neighborhood defined by the kernel and the given mask.
 *
 * @tparam dtype The data type of the image.
 * @param image Input image (marker image).
 * @param mask Mask image.
 * @param output Output image.
 * @param xsize Width of the image.
 * @param ysize Height of the image.
 * @param zsize Depth of the image.
 * @param padding_bottom Padding added at the bottom in the z-dimension.
 * @param padding_top Padding added at the top in the z-dimension.
 * @param centerIdx X-coordinate of the pixel being processed.
 * @param centerIdy Y-coordinate of the pixel being processed.
 * @param centerIdz Z-coordinate of the pixel being processed.
 * @param kernel_xsize Kernel size in the x-dimension.
 * @param kernel_ysize Kernel size in the y-dimension.
 * @param kernel_zsize Kernel size in the z-dimension.
 * @param operation Morphological operation (EROSION or DILATION).
 * 
 * @note This implementation is based on the morphological operations 
 *       described in "Digital Image Processing, 4th Edition" by R.C. Gonzalez and R.E. Woods, 
 *       particularly in Chapter 9 (Morphological Image Processing), Section 9.6, 
 *       on pages 667-668, adapted for grayscale images.
 * @see R.C. Gonzalez, R.E. Woods, "Digital Image Processing," 4th Edition, Pearson, 2018.
 */
template <typename dtype>
CUDA_HOSTDEV void geodesic_morph_grayscale_pixel(dtype* image, dtype* mask, dtype* output,
                                                 const int xsize, const int ysize, const int zsize,
                                                 const int padding_bottom, const int padding_top,
                                                 int centerIdx, int centerIdy, int centerIdz,
                                                 int kernel_xsize, int kernel_ysize,
                                                 int kernel_zsize, MorphOp operation) {
  dtype* im = image;

  // Initialize auxiliary value with the central pixel
  size_t centerIndex = static_cast<size_t>(centerIdz) * ysize * xsize + 
                       static_cast<size_t>(centerIdy) * xsize + 
                       static_cast<size_t>(centerIdx);

  dtype aux = im[centerIndex];

  int startIdx = centerIdx - kernel_xsize / 2;
  int startIdy = centerIdy - kernel_ysize / 2;
  int startIdz = centerIdz - kernel_zsize / 2;

  size_t index;
  int imageIdx, imageIdy, imageIdz;

  // Erosion/dilation operation
  for (int iz = 0; iz < kernel_zsize; iz++) {
    for (int iy = 0; iy < kernel_ysize; iy++) {
      for (int ix = 0; ix < kernel_xsize; ix++) {

        imageIdx = startIdx + ix;
        imageIdy = startIdy + iy;
        imageIdz = startIdz + iz;

        // Ignore out of bounds pixels
        if (imageIdx < 0 || imageIdx > xsize - 1 || imageIdy < 0 || imageIdy > ysize - 1 ||
            imageIdz < -padding_bottom || imageIdz > zsize + padding_top - 1) {
        }

        else {
          index = static_cast<size_t>(imageIdz) * xsize * ysize + 
                  static_cast<size_t>(imageIdy) * xsize + 
                  static_cast<size_t>(imageIdx);

          if (operation == EROSION) {
            aux = (im[index] < aux) ? im[index] : aux;  // Erosion: aux is the min value
          } else {
            aux = (im[index] > aux) ? im[index] : aux;  // Dilation: aux is  the max value
          }
        }
      }
    }
  }

  // Point-wise maximun/minimun operation
  if (operation == EROSION) {
    output[centerIndex] =
        (aux > mask[centerIndex]) ? aux : mask[centerIndex];  // Erosion: output is the max value
  } else {
    output[centerIndex] =
        (aux < mask[centerIndex]) ? aux : mask[centerIndex];  // Dilation: output is the min value
  }
}
template CUDA_HOSTDEV void geodesic_morph_grayscale_pixel<int>(int*, int*, int*, const int,
                                                               const int, const int, const int,
                                                               const int, int, int, int, int, int,
                                                               int, MorphOp);
template CUDA_HOSTDEV void geodesic_morph_grayscale_pixel<unsigned int>(
    unsigned int*, unsigned int*, unsigned int*, const int, const int, const int, const int,
    const int, int, int, int, int, int, int, MorphOp);
template CUDA_HOSTDEV void geodesic_morph_grayscale_pixel<float>(float*, float*, float*, const int,
                                                                 const int, const int, const int,
                                                                 const int, int, int, int, int, int,
                                                                 int, MorphOp);
/**
 * @brief CUDA kernel for geodesic grayscale erosion/dilation on an entire image.
 *
 * @tparam dtype The data type of the image.
 * @param deviceImage Input image on the GPU.
 * @param deviceMask Mask image on the GPU.
 * @param deviceOutput Output image on the GPU.
 * @param xsize Width of the image.
 * @param ysize Height of the image.
 * @param zsize Depth of the image.
 * @param padding_bottom Padding at the bottom in the z-dimension.
 * @param padding_top Padding at the top in the z-dimension.
 * @param kernel_xsize Kernel size in x-dimension.
 * @param kernel_ysize Kernel size in y-dimension.
 * @param kernel_zsize Kernel size in z-dimension.
 * @param operation Morphological operation (EROSION or DILATION).
 * 
 * @note This implementation is based on the morphological operations 
 *       described in "Digital Image Processing, 4th Edition" by R.C. Gonzalez and R.E. Woods, 
 *       particularly in Chapter 9 (Morphological Image Processing), Section 9.6, 
 *       on pages 667-668, adapted for grayscale images.
 * @see geodesic_morph_grayscale_pixel()
 */
template <typename dtype>
__global__ void geodesic_morph_grayscale_kernel(dtype* deviceImage, dtype* deviceMask,
                                                dtype* deviceOutput, const int xsize,
                                                const int ysize, const int zsize,
                                                const int padding_bottom, const int padding_top,
                                                int kernel_xsize, int kernel_ysize,
                                                int kernel_zsize, MorphOp operation) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int idy = threadIdx.y + blockIdx.y * blockDim.y;
  int idz = threadIdx.z + blockIdx.z * blockDim.z;

  if (idx < xsize && idy < ysize && idz < zsize) {
    geodesic_morph_grayscale_pixel(deviceImage, deviceMask, deviceOutput, xsize, ysize, zsize,
                                   padding_bottom, padding_top, idx, idy, idz, kernel_xsize,
                                   kernel_ysize, kernel_zsize, operation);
  }
}
template __global__ void geodesic_morph_grayscale_kernel<int>(int*, int*, int*, const int,
                                                              const int, const int, const int,
                                                              const int, int, int, int, MorphOp);
template __global__ void geodesic_morph_grayscale_kernel<unsigned int>(unsigned int*, unsigned int*,
                                                                       unsigned int*, const int,
                                                                       const int, const int,
                                                                       const int, const int, int,
                                                                       int, int, MorphOp);
template __global__ void geodesic_morph_grayscale_kernel<float>(float*, float*, float*, const int,
                                                                const int, const int, const int,
                                                                const int, int, int, int, MorphOp);

template <typename dtype>
void geodesic_morph_grayscale(dtype* deviceImage, dtype* deviceMask, dtype* deviceOutput,
                              const int xsize, const int ysize, const int zsize,
                              const int flag_verbose, const int padding_bottom,
                              const int padding_top, MorphOp operation) {

  //define connectivity kernel size for images of any dimension
  int kernel_xsize = (xsize > 2) ? 3 : xsize;
  int kernel_ysize = (ysize > 2) ? 3 : ysize;
  int kernel_zsize = (zsize > 2) ? 3 : zsize;

  //set up execution configuratio
  dim3 block(BLOCK_3D, BLOCK_3D, BLOCK_3D);
  if (zsize == 1)
    block = dim3(BLOCK_2D, BLOCK_2D, 1);

  dim3 grid((xsize + block.x - 1) / block.x, (ysize + block.y - 1) / block.y,
            (zsize + block.z - 1) / block.z);

  // check grid and block dimension from host side
  if (flag_verbose) {
    printf("grid.x %d grid.y %d grid.z %d\n", grid.x, grid.y, grid.z);
    printf("block.x %d block.y %d block.z %d\n", block.x, block.y, block.z);
  }

  // device erosion/dialation
  geodesic_morph_grayscale_kernel<<<grid, block>>>(
      deviceImage, deviceMask, deviceOutput, xsize, ysize, zsize, padding_bottom, padding_top,
      kernel_xsize, kernel_ysize, kernel_zsize, operation);
  hipDeviceSynchronize();  //assures all gpu threads are fineshed
}
template void geodesic_morph_grayscale<int>(int*, int*, int*, const int, const int, const int,
                                            const int, const int, const int, MorphOp);
template void geodesic_morph_grayscale<unsigned int>(unsigned int*, unsigned int*, unsigned int*,
                                                     const int, const int, const int, const int,
                                                     const int, const int, MorphOp);
template void geodesic_morph_grayscale<float>(float*, float*, float*, const int, const int,
                                              const int, const int, const int, const int, MorphOp);

template <typename dtype>
void geodesic_morph_grayscale_on_device(dtype* hostImage, dtype* hostMask, dtype* hostOutput,
                                        const int xsize, const int ysize, const int zsize,
                                        const int flag_verbose, const int padding_bottom,
                                        const int padding_top, MorphOp operation) {
  // set input dimension
  size_t size = static_cast<size_t>(xsize) * ysize * zsize;
  size_t nBytes = size * sizeof(dtype);
  size_t nBytes_padding = xsize * ysize * (padding_bottom + padding_top) * sizeof(dtype);
  size_t nBytes_input = nBytes + nBytes_padding;

  // malloc device global memory
  dtype *deviceImage, *deviceOutput, *deviceMask, *i_deviceImage, *i_hostImage, *i_deviceMask,
      *i_hostMask;
  CHECK(hipMalloc((dtype**)&i_deviceImage, nBytes_input));
  CHECK(hipMalloc((dtype**)&i_deviceMask, nBytes_input));
  CHECK(hipMalloc((dtype**)&deviceOutput, nBytes));

  // transfer input and mask + padding
  i_hostImage = hostImage - padding_bottom * xsize * ysize;
  i_hostMask = hostMask - padding_bottom * xsize * ysize;

  CHECK(hipMemcpy(i_deviceImage, i_hostImage, nBytes_input, hipMemcpyHostToDevice));
  CHECK(hipMemcpy(i_deviceMask, i_hostMask, nBytes_input, hipMemcpyHostToDevice));

  deviceImage = i_deviceImage + padding_bottom * xsize * ysize;
  deviceMask = i_deviceMask + padding_bottom * xsize * ysize;

  // device erosion/dialation
  geodesic_morph_grayscale(deviceImage, deviceMask, deviceOutput, xsize, ysize, zsize, flag_verbose,
                           padding_bottom, padding_top, operation);

  // transfer data from the device to the host
  CHECK(hipMemcpy(hostOutput, deviceOutput, nBytes, hipMemcpyDeviceToHost));

  // free host memorys
  hipFree(i_deviceImage);
  hipFree(i_deviceMask);
  hipFree(deviceOutput);
}
template void geodesic_morph_grayscale_on_device<int>(int*, int*, int*, const int, const int,
                                                      const int, const int, const int, const int,
                                                      MorphOp);
template void geodesic_morph_grayscale_on_device<unsigned int>(unsigned int*, unsigned int*,
                                                               unsigned int*, const int, const int,
                                                               const int, const int, const int,
                                                               const int, MorphOp);
template void geodesic_morph_grayscale_on_device<float>(float*, float*, float*, const int,
                                                        const int, const int, const int, const int,
                                                        const int, MorphOp);

template <typename dtype>
void geodesic_morph_grayscale_on_host(dtype* hostImage, dtype* hostMask, dtype* hostOutput,
                                      const int xsize, const int ysize, const int zsize,
                                      MorphOp operation) {

  //define connectivity kernel size for images of any dimension
  int kernel_xsize = (xsize > 2) ? 3 : xsize;
  int kernel_ysize = (ysize > 2) ? 3 : ysize;
  int kernel_zsize = (zsize > 2) ? 3 : zsize;

  for (int idz = 0; idz < zsize; idz++) {
    for (int idy = 0; idy < ysize; idy++) {
      for (int idx = 0; idx < xsize; idx++) {

        geodesic_morph_grayscale_pixel(hostImage, hostMask, hostOutput, xsize, ysize, zsize, 0, 0,
                                       idx, idy, idz, kernel_xsize, kernel_ysize, kernel_zsize,
                                       operation);
      }
    }
  }  // slide over image
}
template void geodesic_morph_grayscale_on_host<int>(int*, int*, int*, const int, const int,
                                                    const int, MorphOp);
template void geodesic_morph_grayscale_on_host<unsigned int>(unsigned int*, unsigned int*,
                                                             unsigned int*, const int, const int,
                                                             const int, MorphOp);
template void geodesic_morph_grayscale_on_host<float>(float*, float*, float*, const int, const int,
                                                      const int, MorphOp);
